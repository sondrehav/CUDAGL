#include "hip/hip_runtime.h"
#include "julia_set.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 128

#define CHND(x) \
	if(x != hipSuccess) {\
		fprintf(stderr, "CUDA failed at line %d in file %s!", __LINE__, __FILE__);\
		exit(-1);\
	}\

__global__ void add(float *a, float *b, float *out)
{
	int thread_id = blockIdx.x;
	if (thread_id < N)
	{
		out[thread_id] = a[thread_id] + b[thread_id];
	}
}

int test()
{
	srand(time(NULL));
	float a[N], b[N], out[N];
	float* dev_a, *dev_b, *dev_out;

	CHND(hipMalloc((void**)&dev_a, N * sizeof(float)));
	CHND(hipMalloc((void**)&dev_b, N * sizeof(float)));
	CHND(hipMalloc((void**)&dev_out, N * sizeof(float)));

	for (int i = 0; i < N; i++)
	{
		a[i] = (float)rand() / (float)RAND_MAX;
		b[i] = (float)rand() / (float)RAND_MAX;
	}

	CHND(hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice));
	CHND(hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice));

	add<<<N, 1>>>(dev_a, dev_b, dev_out);

	CHND(hipMemcpy(out, dev_out, N * sizeof(float), hipMemcpyDeviceToHost));

	printf("[ ");
	for (int i = 0; i < N; i++)
	{
		printf("%3.3f, ", out[i]);
	}
	printf(" ]\n");

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_out);

	return 0;

}
